#include "hip/hip_runtime.h"
#include "cuda_LinSolver.h"

#define BS_X 32
#define BS_Y 32

// k is the current row being used to reduce the rest
__global__ void kernMatReduce(int n, float* gMat, float* iMat, int k) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;  // Row index
	int j = blockDim.y * blockIdx.y + threadIdx.y;  // Column index

	// keep in matrix bounds
	// matrix always square
	// extra column for iMat
	if (i >= n || j > n) return;
	if (i == k) return; // skip reference row

	int ref_idx = k * n;
	// error, need to return somehow?
	if (gMat[ref_idx + k] == 0) return;

	int idx = i * n;

	float ratio = gMat[idx + k] / gMat[ref_idx + k];

	if (j == n) {
		iMat[i] -= ratio * iMat[k];
		return;
	}

	gMat[idx + j] -= ratio * gMat[ref_idx + j];
}

__global__ void kernPlugKnownV(int n, float* gMat, float* iMat, float* vMat) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;  // Row index
	int j = blockDim.y * blockIdx.y + threadIdx.y;  // Column index

	if (i >= n || j >= n) return;
	
	float v = vMat[j];
	float g = gMat[i * n + j];
	if (v != 0.0f && g != 0.0f) {
		float c = -g * v;
		atomicAdd(iMat + i, c);
	}
}

__global__ void kernMatSolve(int n, float* gMat, float* iMat, float* vMat) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;  // Row index

	// keep in matrix bounds
	// matrix always square
	if (i >= n) return;
	if (vMat[i] != 0.0f) return;

	// error?
	if (gMat[i * n + i] == 0) return;

	float v = iMat[i] / gMat[i * n + i];
	vMat[i] = v;
}

void gpuMatReduce(int n, float* dev_gMat, float* dev_iMat) {

	int numBlocks = ceil(float(n) / 32.0f);

	dim3 numBlocks3D = dim3(numBlocks, numBlocks, 1);
	dim3 blockSize = dim3(BS_X, BS_Y, 1);

	
	for (int i = 0; i < n; i++) {
		kernMatReduce << < numBlocks3D, blockSize >> > (n, dev_gMat, dev_iMat, i);
	}
	checkCUDAError("Reduction Failure!\n");
}

void gpuMatSolve(int n, float** gMat, float* iMat, float* vMat) {

	float* dev_gMat = NULL;
	float* dev_vMat = NULL;
	float* dev_iMat = NULL;

	//setupDevMats(n, gMat, dev_gMat, iMat, dev_iMat, dev_vMat);
	// alloc device memory
	hipMalloc((void**)&dev_iMat, n * sizeof(float));
	hipMalloc((void**)&dev_gMat, n * n * sizeof(float));
	hipMalloc((void**)&dev_vMat, n * sizeof(float));
	checkCUDAError("Malloc Failure!\n");

	// copy host to device
	for (int i = 0; i < n; i++) {
		hipMemcpy(dev_gMat + i*n, gMat[i], n * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("Host gMat MemCpy Failure!\n");
	}
	hipMemcpy(dev_iMat, iMat, n * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("Host iMat MemCpy Failure!\n");
	hipMemcpy(dev_vMat, vMat, n * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("Host vMat MemCpy Failure!\n");


	int numBlocks = ceil(float(n) / BS_X);

	dim3 numBlocks3D = dim3(numBlocks, numBlocks, 1);
	dim3 blockSize = dim3(BS_X, BS_Y, 1);


	for (int i = 0; i < n; i++) {
		kernMatReduce << < numBlocks3D, blockSize >> > (n, dev_gMat, dev_iMat, i);

		checkCUDAError("Reduction Failure!\n");
	}
	hipDeviceSynchronize();

	kernPlugKnownV << < numBlocks3D, blockSize >> > (n, dev_gMat, dev_iMat, dev_vMat);

	kernMatSolve<<<numBlocks, BS_X>>>(n, dev_gMat, dev_iMat, dev_vMat);
	checkCUDAError("Solution Failure!\n");

	hipDeviceSynchronize();

	//copyDevMats(n, gMat, dev_gMat, iMat, dev_iMat, vMat, dev_vMat);
	for (int i = 0; i < n; i++) {
		hipMemcpy(gMat[i], dev_gMat + i*n, n * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("Device gMat MemCpy Failure!\n");
	}

	hipMemcpy(vMat, dev_vMat, n * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("Device vMat MemCpy Failure!\n");

	hipMemcpy(iMat, dev_iMat, n * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("Device iMat MemCpy Failure!\n");

	cleanDevMats(dev_gMat, dev_iMat, dev_vMat);
}