#include "hip/hip_runtime.h"
#include "cuda_kcl.h"

// Parallelize R, IDC, and VCCS list by element
__global__ void kernDCPassiveMat(int n, int n_nodes, CUDA_Elem* passives, float* gMat, float* iMat) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index

	if (idx >= n) return;
	
	CUDA_Elem* e = passives + idx;
	char type = e->type;

	if (type != 'R' && type != 'I' && type != 'G') return;
	
	int a = e->nodes[0] - 1;
	int b = e->nodes[1] - 1;

	
	float val = e->params[0];


	if (type == 'R') {
		val = 1.0f / val;
	}

	// If it's shorted, no contribution
	if (a == b ) return;

	// DC Current Source
	if (type == 'I') {
		if (a >= 0) atomicAdd(iMat + a, -val);
		if (b >= 0) atomicAdd(iMat + b, val);
		return;
	}

	int c = (type == 'G') ? e->nodes[2] : a;
	int d = (type == 'G') ? e->nodes[3] : b;

	if (c == d) return;

	// Resistor or VCCS
	if (a >= 0 && c >= 0) atomicAdd(gMat + (a * n_nodes + c), val);
	if (b >= 0 && d >= 0) atomicAdd(gMat + (b * n_nodes + d), val);

	if (b >= 0 && a >= 0 && d >= 0 && c >= 0) {
		atomicAdd(gMat + (a * n_nodes + d), -val);
		atomicAdd(gMat + (b * n_nodes + c), -val);
	}
	
}

__global__ void kernTranPassMat(int n, int n_nodes, CUDA_Elem* passives, float* gMat, float* iMat, float* vPrev, float h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index

	if (idx >= n) return;

	CUDA_Elem* e = passives + idx;
	char type = e->type;

	if (type != 'R' && type != 'I' && type != 'G' && type != 'C') return;

	int a = e->nodes[0] - 1;
	int b = e->nodes[1] - 1;


	float val = e->params[0];


	if (type == 'R') {
		val = 1.0f / val;
	}
	if (type == 'C') {
		val = val / h;
	}

	// If it's shorted, no contribution
	if (a == b) return;

	// DC Current Source
	if (type == 'I') {
		if (a >= 0) atomicAdd(iMat + a, -val);
		if (b >= 0) atomicAdd(iMat + b, val);
		return;
	}

	int c = (type == 'G') ? e->nodes[2] : a;
	int d = (type == 'G') ? e->nodes[3] : b;

	if (c == d) return;

	// Resistor or VCCS
	if (a >= 0 && c >= 0) {
		atomicAdd(gMat + (a * n_nodes + c), val);
		if (type == 'C') atomicAdd(iMat + a, val * vPrev[a]);
	}
	if (b >= 0 && d >= 0) {
		atomicAdd(gMat + (b * n_nodes + d), val);
		if (type == 'C') atomicAdd(iMat + b, val * vPrev[b]);
	}

	if (b >= 0 && a >= 0 && d >= 0 && c >= 0) {
		atomicAdd(gMat + (a * n_nodes + d), -val);
		atomicAdd(gMat + (b * n_nodes + c), -val);
		if (type == 'C') {
			atomicAdd(iMat + a, -val * vPrev[b]);
			atomicAdd(iMat + b, -val * vPrev[a]);
		}
	}

}


__global__ void kernVDCtoMat(int n_v, int n_nodes, CUDA_Elem* elems, float* gMat, float* iMat, float* vMat) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index
	if (idx >= n_v) return;
	
	CUDA_Elem* e = elems + idx;
	int n = e->nodes[1] - 1; // pos node
	int p = e->nodes[0] - 1; // neg node

	// shorted
	if (n == p) return;

	float val = e->params[0];

	// negative node grounded,
	// most common case
	if (p >= 0 && n < 0) {

		gMat[p * (n_nodes + 1)] = 1.0f;
		iMat[p] = val;
		vMat[p] = val;
	}
	// positive node grounded
	else if (p < 0 && n >= 0) {

		gMat[n * (n_nodes + 1)] = 1.0f;
		iMat[n] = -val;
		vMat[n] = -val;
	}
	// neither grounded
	else {

		iMat[p] = val;
		gMat[p * n_nodes + n] = -1.0f;
		gMat[p * (n_nodes + 1)] = 1.0f;

		if (vMat[p] != 0.0f) vMat[n] = vMat[p] - val;
		else if (vMat[n] != 0.0f) vMat[p] = vMat[n] + val;
	}

}

__global__ void kernTranVtoMat(int n_v, int n_nodes, float time, CUDA_Elem* elems, float* gMat, float* iMat, float* vMat) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index
	if (idx >= n_v) return;

	CUDA_Elem* V = elems + idx;
	int n = V->nodes[1] - 1; // pos node
	int p = V->nodes[0] - 1; // neg node

	// shorted
	if (n == p) return;

	// Fully DC Source
	if (V->type == 'V') {
		float val = V->params[0];
		if (p >= 0 && n < 0) {
			gMat[p * (n_nodes + 1)] = 1.0f;
			iMat[p] = val;
			vMat[p] = val;
		}
		else if (p < 0 && n >= 0) {
			gMat[n * (n_nodes + 1)] = 1.0f;
			iMat[n] = -val;
			vMat[n] = -val;
		}
		else {
			iMat[p] = val;
			gMat[p * n_nodes + n] = -1.0f;
			gMat[p * (n_nodes + 1)] = 1.0f;
			if (vMat[p] != 0.0f) vMat[n] = vMat[p] - val;
			else if (vMat[n] != 0.0f) vMat[p] = vMat[n] + val;
		}
	}
	else if (V->type == 'P') {
		// Pulse Parameters
		float V1 = V->params[1]; // initial val
		float V2 = V->params[2]; // peak val
		float td = V->params[3]; // initial delay
		float tr = V->params[4]; // rise time
		float tf = V->params[5]; // fall time
		float width = V->params[6]; // pulse width
		float period = V->params[7]; // period, time for one cycle

		// put time in context of current pulse
		float p_time = (time - td) - period * floor((time - td) / period);

		// Calculate voltage value for time instance
		float val;
		if (time < td) val = 0.0f; // time is before pulse start
		else if (p_time < tr) val = V1 + (p_time * (V2 - V1)) / tr; // interpolate value
		else if (p_time < width) val = V2;
		else if (p_time < tf + width) val = V2 - ((p_time - width) * (V2 - V1)) / tf; // interpolate value
		else val = V1;

		if (p >= 0 && n < 0) {
			gMat[p * (n_nodes + 1)] = 1.0f;
			iMat[p] = val;
			vMat[p] = val;
		}
		else if (p < 0 && n >= 0) {
			gMat[n * (n_nodes + 1)] = 1.0f;
			iMat[n] = -val;
			vMat[n] = -val;
		}
		else {
			iMat[p] = val;
			gMat[p * n_nodes + n] = -1.0f;
			gMat[p * (n_nodes + 1)] = 1.0f;
			if (vMat[p] != 0.0f) vMat[n] = vMat[p] - val;
			else if (vMat[n] != 0.0f) vMat[p] = vMat[n] + val;
		}
	}
}


__global__ void kernelMOStoMat(int n, int n_nodes, CUDA_Elem* elems, Model* models, float* gMat, float* iMat, float* vGuess) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index
	if (idx >= n) return;

	CUDA_Elem* T = elems + idx;
	Model* M = models + T->model;

	char type = M->type;

	float I;
	float g;

	int n_d = T->nodes[0] - 1;
	int n_g = T->nodes[1] - 1;
	int n_s = T->nodes[2] - 1;
	//int n_b = T->nodes[3] - 1;

	// Load guessed node voltages
	float Vg = 0.0f;
	if (n_g >= 0) Vg = vGuess[n_g];
	float Vs = 0.0f;
	if (n_s >= 0) Vs = vGuess[n_s];
	float Vd = 0.0f;
	if (n_d >= 0) Vd = vGuess[n_d];

	// Switch S and D depending on voltage seen
	if ((Vs > Vd && type == 'n') || (Vs < Vd && type == 'p')) {
		n_s = T->nodes[0] - 1;
		Vs = 0.0f;
		if (n_s >= 0) Vs = vGuess[n_s];

		n_d = T->nodes[2] - 1;
		Vd = 0.0f;
		if (n_d >= 0) Vd = vGuess[n_d];
	}

	float vth = M->vt0;
	float Vov = Vg - Vs - vth;

	float Vds = Vd - Vs;

	// Ideally no current flows through channel for Vds = 0
	//if (Vds == 0.0f) return;

	//Subthreshold, not yet handled
	if ((Vov <= 0 && type == 'n') || (Vov >= 0 && type == 'p')) return;

	// "Constants"
	float L = T->params[0];
	float W = T->params[1];
	float Cox = (M->epsrox * PERMITTIVITY / (M->tox * 100.f));

	float k = (W / L) * M->u0 * Cox;

	float CLM = M->pclm * Vov;

	if (type == 'p') {
		k = -k;
		CLM = -CLM;
	}

	// Saturation, usually desired case
	if ((Vds > Vov && type == 'n') || (Vds < Vov && type == 'p')) {

		g = 0.5f * k * Vov;
		I = g * (1 - CLM) * vth;

		if (n_d >= 0) {
			atomicAdd(iMat + n_d, I);
			atomicAdd(gMat + (n_d * (n_nodes + 1)), g * CLM);
			if (n_g >= 0) atomicAdd(gMat + (n_d * n_nodes + n_g), g * (1 - CLM));
			if (n_s >= 0) atomicAdd(gMat + (n_d * n_nodes + n_s), -g);
		}
		if (n_s >= 0) {
			atomicAdd(iMat + n_s, -I);
			atomicAdd(gMat + (n_s * (n_nodes + 1)), g);
			if (n_g >= 0) atomicAdd(gMat + (n_s * n_nodes + n_g), -g * (1 - CLM));
			if (n_d >= 0) atomicAdd(gMat + (n_s * n_nodes + n_d), -g * CLM);
		}
	}
	// "linear" region
	else {
		g = k * Vov;
		I = k * 0.5 * (Vds * Vds);

		if (n_d >= 0) {
			atomicAdd(gMat + (n_d * (n_nodes + 1)), g);
			atomicAdd(iMat + n_d, I);
			if (n_s >= 0) atomicAdd(gMat + (n_d * n_nodes + n_s), -g);
		}
		if (n_s >= 0) {
			atomicAdd(gMat + (n_s * (n_nodes + 1)), g);
			atomicAdd(iMat + n_s, -I);
			if (n_d >= 0) atomicAdd(gMat + (n_s * n_nodes + n_d), -g);
		}
	}
}

__global__ void kernelTranMOStoMat(int n, int n_nodes, CUDA_Elem* elems, Model* models, float* gMat, float* iMat, float* vGuess, float* vPrev, float h) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index
	if (idx >= n) return;

	CUDA_Elem* T = elems + idx;
	Model* M = models + T->model;
	char type = M->type;

	float I;
	float g;

	int n_d = T->nodes[0] - 1;
	int n_g = T->nodes[1] - 1;
	int n_s = T->nodes[2] - 1;
	int n_b = T->nodes[3] - 1;

	// Load guessed node voltages
	float Vg = 0.0f;
	if (n_g >= 0) Vg = vGuess[n_g];
	float Vs = 0.0f;
	if (n_s >= 0) Vs = vGuess[n_s];
	float Vd = 0.0f;
	if (n_d >= 0) Vd = vGuess[n_d];

	if ((Vs > Vd && type == 'n') || (Vs < Vd && type == 'p')) {
		n_s = T->nodes[0] - 1;
		Vs = 0.0f;
		if (n_s >= 0) Vs = vGuess[n_s];

		n_d = T->nodes[2] - 1;
		Vd = 0.0f;
		if (n_d >= 0) Vd = vGuess[n_d];
	}

	float vth = M->vt0;
	float Vov = Vg - Vs - vth;
	float Vds = Vd - Vs;

	// "Constants"
	float L = T->params[0];
	float W = T->params[1];
	float Cox = (M->epsrox * PERMITTIVITY / (M->tox * 100.f));

	float k = (W / L) * M->u0 * Cox;

	float CLM = M->pclm * Vov;

	if (type == 'p') {
		k = -k;
		CLM = -CLM;
	}

	float Cgcb = Cox * 1e-4 * W * L;

	if ((Vov <= 0 && type == 'n') || (Vov >= 0 && type == 'p')) {
		// Cgcb
		float G = Cgcb / h;
		if (n_g >= 0) {
			atomicAdd(gMat + (n_g * (n_nodes + 1)), G);
			atomicAdd(iMat + n_g, G * vPrev[n_g]);
		}
		if (n_b >= 0) {
			atomicAdd(gMat + (n_b * (n_nodes + 1)), G);
			atomicAdd(iMat + n_b, G * vPrev[n_b]);
		}
		if (n_g >= 0 && n_b >= 0) {
			atomicAdd(gMat + (n_g * n_nodes + n_b), -G);
			atomicAdd(gMat + (n_b * n_nodes + n_g), -G);
			atomicAdd(iMat + n_g, -G * vPrev[n_b]);
			atomicAdd(iMat + n_b, -G * vPrev[n_g]);
		}
		return;
	}

	// Saturation, usually desired case
	if ((Vds > Vov && type == 'n') || (Vds < Vov && type == 'p')) {
		float G = (2.0 / 3.0) * Cgcb / h;

		g = 0.5f * k * Vov;
		I = g * (1 - CLM) * vth;

		if (n_g >= 0) {
			atomicAdd(gMat + (n_g * n_nodes + n_g), G);
			atomicAdd(iMat + n_g, G * vPrev[n_g]);
		}
		if (n_d >= 0) {
			atomicAdd(iMat + n_d, I);
			atomicAdd(gMat + (n_d * (n_nodes + 1)), g * CLM);
			if (n_g >= 0) atomicAdd(gMat + (n_d * n_nodes + n_g), g * (1 - CLM));
			if (n_s >= 0) atomicAdd(gMat + (n_d * n_nodes + n_s), -g);
		}
		if (n_s >= 0) {
			atomicAdd(iMat + n_s, -I + G * vPrev[n_s]);
			atomicAdd(gMat + (n_s * (n_nodes + 1)), g + G);
			if (n_g >= 0) {
				atomicAdd(gMat + (n_s * n_nodes + n_g), -g * (1 - CLM) - G);
				atomicAdd(gMat + (n_g * n_nodes + n_s), -G);
				atomicAdd(iMat + n_g, -G * vPrev[n_s]);
				atomicAdd(iMat + n_s, -G * vPrev[n_g]);
			}
			if (n_d >= 0) atomicAdd(gMat + (n_s * n_nodes + n_d), -g * CLM);
		}
	}
	// "linear" region
	else {
		float ratio = (Vd - Vs) / Vov;
		float Gs = (0.5f + ratio / 6.0f) * Cgcb / h;
		float Gd = 0.5f * (1 - ratio) * Cgcb / h;

		g = k * Vov;
		I = k * 0.5 * (Vds * Vds);

		if (n_g >= 0) {
			atomicAdd(gMat + (n_g * (n_nodes + 1)), Gs + Gd);
			atomicAdd(iMat + n_g, (Gs + Gd) * vPrev[n_g]);
			if (n_s >= 0) {
				atomicAdd(gMat + (n_g * n_nodes + n_s), -Gs);
				atomicAdd(gMat + (n_s * n_nodes + n_g), -Gs);
				atomicAdd(iMat + n_g, -Gs * vPrev[n_s]);
				atomicAdd(iMat + n_s, -Gs * vPrev[n_g]);
			}
			if (n_d >= 0) {
				atomicAdd(gMat + (n_g * n_nodes + n_d), -Gd);
				atomicAdd(gMat + (n_d * n_nodes + n_g), -Gd);
				atomicAdd(iMat + n_g, -Gd * vPrev[n_d]);
				atomicAdd(iMat + n_d, -Gd * vPrev[n_g]);
			}
		}
		if (n_d >= 0) {
			atomicAdd(gMat + (n_d * (n_nodes + 1)), g + Gd);
			atomicAdd(iMat + n_d, I + (Gd * vPrev[n_d]));
			if (n_s >= 0) atomicAdd(gMat + (n_d * n_nodes + n_s), -g);
		}
		if (n_s >= 0) {
			atomicAdd(gMat + (n_s * (n_nodes + 1)), g + Gs);
			atomicAdd(iMat + n_s, -I + (Gs * vPrev[n_s]));
			if (n_d >= 0) atomicAdd(gMat + (n_s * n_nodes + n_d), -g);
		}
	}
}


// First part of setting up Voltage sources in matrices
__global__ void kernelAddandZero(int n, float* gMat_d, float* gMat_s, float* i_d, float* i_s) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index
	if (idx > n) return;

	if (idx == n) {
		if (i_d != NULL) i_d[0] += i_s[0];
		i_s[0] = 0.0f;
		return;
	}

	if (gMat_d != NULL) gMat_d[idx] += gMat_s[idx];
	gMat_s[idx] = 0.0f;

}

// Populates Matrices on GPU side
// use CPU netlist for CPU operations
void gpuNetlistToMatTest(CUDA_Net* dev_net, Netlist* netlist, float** gMat, float* iMat, float* vMat) {
	float* dev_gMat = NULL;
	float* dev_iMat = NULL;
	float* dev_vMat = NULL;

	int n = dev_net->n_nodes;
	if (n == 0) return;
	

	// alloc device memory
	hipMalloc((void**)&dev_iMat, n * sizeof(float));
	hipMalloc((void**)&dev_vMat, n * sizeof(float));
	hipMalloc((void**)&dev_gMat, n * n * sizeof(float));
	checkCUDAError("Malloc Failure!\n");

	hipMemset(dev_iMat, 0, n * sizeof(float));
	hipMemset(dev_vMat, 0, n * sizeof(float));
	hipMemset(dev_gMat, 0, n * n * sizeof(float));
	checkCUDAError("Memset Failure!\n");

	int n_passive = dev_net->n_passive;

	int numBlocks = ceil(float(n_passive) / BS_1D);

	dim3 numBlocks3D = dim3(numBlocks, 1, 1);
	dim3 blockSize = dim3(BS_1D, 1, 1);

	// Passives

	if (n_passive > 0) kernDCPassiveMat << < numBlocks3D, blockSize >> >(n_passive, n, dev_net->passives, dev_gMat, dev_iMat);
	checkCUDAError("Matrix Gen Kernel Failure!\n");


	// Voltage sources

	int n_vdc = dev_net->n_vdc;
	// For each add copy of the + row to the -, then 0 it
	// Then set the Vp - Vn = VDC equation
	int n_p;
	int n_n;

	if (n_vdc > 0) {
		// We're assuming more nodes than voltage sources, generally correct
		numBlocks = ceil(float(n) / float(BS_1D));
		for (int i = 0; i < n_vdc; i++) {
			n_p = netlist->vdcList[i].nodes[0] - 1;
			n_n = netlist->vdcList[i].nodes[1] - 1;
			if (n_p >= 0 && n_n >= 0) kernelAddandZero << < numBlocks3D, blockSize >> > (n, dev_gMat + n * n_n, dev_gMat + n * n_p, dev_iMat + n_n, dev_iMat + n_p);
			else if (n_p >= 0) kernelAddandZero << < numBlocks3D, blockSize >> > (n, NULL, dev_gMat + n * n_p, NULL, dev_iMat + n_p);
			else if (n_n >= 0) kernelAddandZero << < numBlocks3D, blockSize >> > (n, NULL, dev_gMat + n * n_n, NULL, dev_iMat + n_n);
		}

		numBlocks = ceil(float(n_vdc) / float(BS_1D));
		kernVDCtoMat << < numBlocks3D, blockSize >> >(n_vdc, n, dev_net->vdcList, dev_gMat, dev_iMat, dev_vMat);
	}
	
	copyFromDevMats(n, gMat, dev_gMat, iMat, dev_iMat, vMat, dev_vMat);

	hipFree(dev_gMat);
	hipFree(dev_iMat);
	hipFree(dev_vMat);

	checkCUDAError("Device Matrix Free Failure!\n");
}

void gpuPassiveToMat(CUDA_Net* dev_net, float* dev_gMat, float* dev_iMat) {
	int n = dev_net->n_nodes;
	if (n == 0) return;


	int n_passive = dev_net->n_passive;

	int numBlocks = ceil(float(n_passive) / BS_1D);

	// Passives

	if (n_passive > 0) kernDCPassiveMat << < numBlocks, BS_1D >> >(n_passive, n, dev_net->passives, dev_gMat, dev_iMat);
	checkCUDAError("Matrix Gen Kernel Failure!\n");
}

void gpuPassiveVDCToMat(CUDA_Net* dev_net, Netlist* netlist, float* dev_gMat, float* dev_iMat, float* dev_vMat) {
	int n = dev_net->n_nodes;
	if (n == 0) return;


	int n_passive = dev_net->n_passive;

	int numBlocks = ceil(float(n_passive) / float(BS_1D));

	// Passives

	if (n_passive > 0) kernDCPassiveMat << < numBlocks, BS_1D >> >(n_passive, n, dev_net->passives, dev_gMat, dev_iMat);
	checkCUDAError("Matrix Gen Kernel Failure!\n");


	// Voltage sources

	int n_vdc = dev_net->n_vdc;
	// For each add copy of the + row to the -, then 0 it
	// Then set the Vp - Vn = VDC equation
	int n_p;
	int n_n;

	if (n_vdc > 0) {
		numBlocks = ceil(float(n) / float(BS_1D));
		// We're assuming more nodes than voltage sources, generally correct
		for (int i = 0; i < n_vdc; i++) {
			n_p = netlist->vdcList[i].nodes[0] - 1;
			n_n = netlist->vdcList[i].nodes[1] - 1;
			if (n_p >= 0 && n_n >= 0) kernelAddandZero << < numBlocks, BS_1D >> > (n, dev_gMat + n * n_n, dev_gMat + n * n_p, dev_iMat + n_n, dev_iMat + n_p);
			else if (n_p >= 0) kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_p, NULL, dev_iMat + n_p);
			else if (n_n >= 0) kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_n, NULL, dev_iMat + n_n);
		}
		numBlocks = ceil(float(n_vdc) / float(BS_1D));
		kernVDCtoMat << < numBlocks, BS_1D >> >(n_vdc, n, dev_net->vdcList, dev_gMat, dev_iMat, dev_vMat);
	}

}




void gpuNetlistToMat(CUDA_Net* dev_net, Netlist* netlist, float* dev_gMat, float* dev_iMat, float* dev_vMat, float* dev_vGuess) {
	int n = dev_net->n_nodes;
	if (n == 0) return;


	int n_passive = dev_net->n_passive;

	int numBlocks = ceil(float(n_passive) / float(BS_1D));

	// Passives

	if (n_passive > 0) kernDCPassiveMat << < numBlocks, BS_1D >> >(n_passive, n, dev_net->passives, dev_gMat, dev_iMat);
	checkCUDAError("Matrix Gen Kernel Failure!\n");


	// Transistors

	int n_active = dev_net->n_active;
	if (n_active > 0) {
		numBlocks = ceil(float(n_active) / float(BS_1D));
		kernelMOStoMat<<<numBlocks, BS_1D>>>(n_active, n, dev_net->actives, dev_net->modelList, dev_gMat, dev_iMat, dev_vGuess);
		checkCUDAError("MOS Matrix Kernel Failure!\n");
	}


	// Voltage sources

	int n_vdc = dev_net->n_vdc;
	// For each add copy of the + row to the -, then 0 it
	// Then set the Vp - Vn = VDC equation
	int n_p;
	int n_n;

	if (n_vdc > 0) {
		// We're assuming more nodes than voltage sources, generally correct
		numBlocks = ceil(float(n) / float(BS_1D));
		for (int i = 0; i < n_vdc; i++) {
			n_p = netlist->vdcList[i].nodes[0] - 1;
			n_n = netlist->vdcList[i].nodes[1] - 1;
			if (n_p >= 0 && n_n >= 0) {
				kernelAddandZero << < numBlocks, BS_1D >> > (n, dev_gMat + n * n_n, dev_gMat + n * n_p, dev_iMat + n_n, dev_iMat + n_p);
				checkCUDAError("vdc setup failed!");
			}
			else if (n_p >= 0) {
				kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_p, NULL, dev_iMat + n_p);
				checkCUDAError("vdc setup failed!");
			}
			else if (n_n >= 0) {
				kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_n, NULL, dev_iMat + n_n);
				checkCUDAError("vdc setup failed!");
			}
		}
		numBlocks = ceil(float(n_vdc) / float(BS_1D));
		kernVDCtoMat << < numBlocks, BS_1D >> >(n_vdc, n, dev_net->vdcList, dev_gMat, dev_iMat, dev_vMat);
		checkCUDAError("vdc setup fail!");
	}

}


void gpuTranPassVToMat(CUDA_Net* dev_net, Netlist* netlist, float* dev_gMat, float* dev_iMat, float* dev_vMat, float* dev_vPrev, float time, float h) {
	int n = dev_net->n_nodes;
	if (n == 0) return;
	int n_passive = dev_net->n_passive;
	int numBlocks = ceil(float(n_passive) / float(BS_1D));

	// Passives
	if (n_passive > 0) kernTranPassMat << < numBlocks, BS_1D >> >(n_passive, n, dev_net->passives, dev_gMat, dev_iMat, dev_vPrev, h);
	checkCUDAError("Matrix Gen Kernel Failure!\n");

	// Voltage sources
	int n_vdc = dev_net->n_vdc;
	int n_p;
	int n_n;

	if (n_vdc > 0) {
		numBlocks = ceil(float(n) / float(BS_1D));
		// We're assuming more nodes than voltage sources, generally correct
		for (int i = 0; i < n_vdc; i++) {
			n_p = netlist->vdcList[i].nodes[0] - 1;
			n_n = netlist->vdcList[i].nodes[1] - 1;
			if (n_p >= 0 && n_n >= 0) kernelAddandZero << < numBlocks, BS_1D >> > (n, dev_gMat + n * n_n, dev_gMat + n * n_p, dev_iMat + n_n, dev_iMat + n_p);
			else if (n_p >= 0) kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_p, NULL, dev_iMat + n_p);
			else if (n_n >= 0) kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_n, NULL, dev_iMat + n_n);
		}
		numBlocks = ceil(float(n_vdc) / float(BS_1D));
		kernTranVtoMat << < numBlocks, BS_1D >> >(n_vdc, n, time, dev_net->vdcList, dev_gMat, dev_iMat, dev_vMat);
	}

}

void gpuTranNetToMat(CUDA_Net* dev_net, Netlist* netlist, float* dev_gMat, float* dev_iMat, float* dev_vMat, float* dev_vGuess, float* dev_vPrev, float time, float h) {
	int n = dev_net->n_nodes;
	if (n == 0) return;
	int n_passive = dev_net->n_passive;
	int numBlocks = ceil(float(n_passive) / float(BS_1D));

	// Passives
	if (n_passive > 0) kernTranPassMat << < numBlocks, BS_1D >> >(n_passive, n, dev_net->passives, dev_gMat, dev_iMat, dev_vPrev, h);
	checkCUDAError("Matrix Gen Kernel Failure!\n");

	// Transistors
	int n_active = dev_net->n_active;
	if (n_active > 0) {
		numBlocks = ceil(float(n_active) / float(BS_1D));
		kernelTranMOStoMat << <numBlocks, BS_1D >> >(n_active, n, dev_net->actives, dev_net->modelList, dev_gMat, dev_iMat, dev_vGuess, dev_vPrev, h);
		checkCUDAError("MOS Matrix Kernel Failure!\n");
	}

	// Voltage sources
	int n_vdc = dev_net->n_vdc;
	int n_p;
	int n_n;
	if (n_vdc > 0) {
		// We're assuming more nodes than voltage sources, generally correct
		numBlocks = ceil(float(n) / float(BS_1D));
		for (int i = 0; i < n_vdc; i++) {
			n_p = netlist->vdcList[i].nodes[0] - 1;
			n_n = netlist->vdcList[i].nodes[1] - 1;
			if (n_p >= 0 && n_n >= 0) {
				kernelAddandZero << < numBlocks, BS_1D >> > (n, dev_gMat + n * n_n, dev_gMat + n * n_p, dev_iMat + n_n, dev_iMat + n_p);
				checkCUDAError("vdc setup failed!");
			}
			else if (n_p >= 0) {
				kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_p, NULL, dev_iMat + n_p);
				checkCUDAError("vdc setup failed!");
			}
			else if (n_n >= 0) {
				kernelAddandZero << < numBlocks, BS_1D >> > (n, NULL, dev_gMat + n * n_n, NULL, dev_iMat + n_n);
				checkCUDAError("vdc setup failed!");
			}
		}
		numBlocks = ceil(float(n_vdc) / float(BS_1D));
		kernTranVtoMat << < numBlocks, BS_1D >> >(n_vdc, n, time, dev_net->vdcList, dev_gMat, dev_iMat, dev_vMat);
		checkCUDAError("vdc setup fail!");
	}
}