#include "cuda_setup.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
	exit(EXIT_FAILURE);
}

// Mallocs arrays and copies host data to device
void setupDevMats(int n, float** gMat, float* dev_gMat, float* iMat, float* dev_iMat, float* dev_vMat) {

	// alloc device memory
	hipMalloc((void**)&dev_iMat, n * sizeof(float));
	hipMalloc((void**)&dev_gMat, n * n * sizeof(float));
	hipMalloc((void**)&dev_vMat, n * sizeof(float));
	checkCUDAError("Malloc Failure!\n");

	// copy host to device
	for (int i = 0; i < n; i++) {
		hipMemcpy(dev_gMat + i*n, gMat[i], n * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("Host gMat MemCpy Failure!\n");
	}
	hipMemcpy(dev_iMat, iMat, n * sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("Host iMat MemCpy Failure!\n");

	hipDeviceSynchronize();
}

// copy data to host
void copyDevMats(int n, float** gMat, float* dev_gMat, float* iMat, float* dev_iMat, float* vMat, float* dev_vMat) {

	for (int i = 0; i < n; i++) {
		hipMemcpy(gMat[i], dev_gMat + i*n, n * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("Device gMat MemCpy Failure!\n");
	}

	hipMemcpy(vMat, dev_vMat, n * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("Device vMat MemCpy Failure!\n");

	hipMemcpy(iMat, dev_iMat, n * sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("Device iMat MemCpy Failure!\n");

	hipDeviceSynchronize();
}

void cleanDevMats(float* dev_gMat, float* dev_iMat, float* dev_vMat) {
	hipFree(dev_gMat);
	hipFree(dev_iMat);
	hipFree(dev_vMat);
}