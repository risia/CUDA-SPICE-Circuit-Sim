#include "cuda_setup.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess == err) {
		return;
	}

	fprintf(stderr, "CUDA error");
	if (file) {
		fprintf(stderr, " (%s:%d)", file, line);
	}
	fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
	system("pause");
	exit(EXIT_FAILURE);
}


// Mallocs arrays and copies host data to device
void setupDevMats(int n, float** gMat, float* &dev_gMat, float* iMat, float* &dev_iMat, float* vMat, float* &dev_vMat) {

	// alloc device memory
	hipMalloc((void**)&dev_iMat, n * sizeof(float));
	hipMalloc((void**)&dev_gMat, n * n * sizeof(float));
	hipMalloc((void**)&dev_vMat, n * sizeof(float));
	checkCUDAError("Malloc Failure!\n");

	copyToDevMats(n, gMat, dev_gMat, iMat, dev_iMat, vMat, dev_vMat);
}


// copy data to host
void copyFromDevMats(int n, float** gMat, float* dev_gMat, float* iMat, float* dev_iMat, float* vMat, float* dev_vMat) {
	if (gMat != NULL) {
		for (int i = 0; i < n; i++) {
			hipMemcpy(gMat[i], dev_gMat + i*n, n * sizeof(float), hipMemcpyDeviceToHost);
			checkCUDAError("Device gMat MemCpy Failure!\n");
		}
	}
	if (vMat != NULL) {
		hipMemcpy(vMat, dev_vMat, n * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("Device vMat MemCpy Failure!\n");
	}
	if (iMat != NULL) {
		hipMemcpy(iMat, dev_iMat, n * sizeof(float), hipMemcpyDeviceToHost);
		checkCUDAError("Device iMat MemCpy Failure!\n");
	}
}

void copyToDevMats(int n, float** gMat, float* dev_gMat, float* iMat, float* dev_iMat, float* vMat, float* dev_vMat) {
	if (gMat != NULL) {// && dev_gMat != NULL) {
		for (int i = 0; i < n; i++) {
			hipMemcpy(dev_gMat + i*n, gMat[i], n * sizeof(float), hipMemcpyHostToDevice);
			checkCUDAError("Host gMat MemCpy Failure!\n");
		}
	}
	if (iMat != NULL) {// && dev_iMat != NULL) {
		hipMemcpy(dev_iMat, iMat, n * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("Host iMat MemCpy Failure!\n");
	}
	if (vMat != NULL) {// && dev_vMat != NULL) {
		hipMemcpy(dev_vMat, vMat, n * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("Host vMat MemCpy Failure!\n");
	}
}

void cleanDevMats(float* dev_gMat, float* dev_iMat, float* dev_vMat) {
	hipFree(dev_gMat);
	hipFree(dev_iMat);
	hipFree(dev_vMat);
}