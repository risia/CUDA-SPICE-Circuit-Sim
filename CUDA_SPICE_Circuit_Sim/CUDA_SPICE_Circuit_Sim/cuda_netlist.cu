#include "hip/hip_runtime.h"
#include "cuda_netlist.h"

__global__ void kernElementPointers(int n, CUDA_Elem* elemList, int** nodeLists, float** paramLists, int* modelIdx, char* type) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index

	if (idx >= n) return;

	elemList[idx].type = type[idx];
	elemList[idx].model = modelIdx[idx];
	elemList[idx].nodes = nodeLists[idx];
	elemList[idx].params = paramLists[idx];
}

int findModelN(Model** modelList, char* name, int n) {
	int i = 0;
	for (i = 0; i < n; i++) {
		if (strcmp(name, modelList[i]->name) == 0) return i;
	}
	return -1;
}
/*
__global__ void kernFreeArrays(int n_max, CUDA_Net* netlist) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; // element index

	if (idx >= n_max) return;

	int n = netlist->n_passive;
	if (idx < n && n > 0) {
		hipFree(netlist->passives[idx].nodes);
		hipFree(netlist->passives[idx].params);
	}
	n = netlist->n_active;
	if (idx < n && n > 0) {
		hipFree(netlist->actives[idx].nodes);
		hipFree(netlist->actives[idx].params);
	}
	n = netlist->n_vdc;
	if (idx < n && n > 0) {
		hipFree(netlist->vdcList[idx].nodes);
		hipFree(netlist->vdcList[idx].params);
	}


	if (idx == 0) {
		hipFree(netlist->modelList);
		hipFree(netlist->passives);
		hipFree(netlist->vdcList);
		hipFree(netlist->actives);
		hipFree(netlist->modelList);
	}
}
*/

void gpuElementCpy(int n, Element* elemList, CUDA_Elem* dev_elemList, Model** modelList, int num_models) {
	
	int model = 0;
	int n_nodes;
	int n_params;

	int** dev_nodeLists;
	float** dev_paramLists;
	int* dev_modelIdxs;
	char* dev_types;

	int* dev_nList;
	float* dev_pList;

	hipMalloc((void**)&dev_nodeLists, n * sizeof(int*));
	hipMalloc((void**)&dev_paramLists, n * sizeof(float*));
	hipMalloc((void**)&dev_modelIdxs, n * sizeof(int));
	hipMalloc((void**)&dev_types, n * sizeof(char));

	checkCUDAError("Temp Pointer Array Malloc Failure!\n");

	// setup element data arrays and arrays to store their pointers
	for (int i = 0; i < n; i++) {
		n_nodes = elemList[i].nodes.size();
		n_params = elemList[i].params.size();

		// allocate node list and parameter list

		hipMalloc((void**)&(dev_nList), n_nodes * sizeof(int));
		hipMalloc((void**)&(dev_pList), n_params * sizeof(float));
		checkCUDAError("Element Arrays Malloc Failure!\n");

		// copy from host
		hipMemcpy(dev_nList, elemList[i].nodes.data(), n_nodes * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_pList, elemList[i].params.data(), n_params * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dev_types, &(elemList[i].type), sizeof(char), hipMemcpyHostToDevice);
		checkCUDAError("Element Arrays Copy Failure!\n");

		// copy pointer to array
		hipMemcpy((dev_nodeLists + i), &dev_nList, sizeof(int*), hipMemcpyHostToDevice);
		hipMemcpy((dev_paramLists + i), &dev_pList, sizeof(float*), hipMemcpyHostToDevice);

		// find which model if MOSFET
		if (modelList != NULL) {
			model = findModelN(modelList, elemList[i].model->name, num_models);
			hipMemcpy(dev_modelIdxs, &model, sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("Model Index Copy Failure!\n");
		}
	}

	// Copy Pointers to Structs
	int numBlocks = ceil(float(n) / 64.0f);

	dim3 numBlocks3D = dim3(numBlocks, 1, 1);
	dim3 blockSize = dim3(64, 1, 1);

	kernElementPointers << < numBlocks3D, blockSize >> >(n, dev_elemList, dev_nodeLists, dev_paramLists, dev_modelIdxs, dev_types);
	checkCUDAError("Element Arrays Pointer Copy Failure!\n");

	hipFree(dev_paramLists);
	hipFree(dev_nodeLists);
	hipFree(dev_modelIdxs);
	hipFree(dev_types);

	checkCUDAError("Temp Pointer Array Free Failure!\n");
}

// Copy netlist to GPU
void gpuNetlist(Netlist* netlist, CUDA_Net* dev_net) {
	// allocate device netlist arrays
	dev_net->actives = NULL;
	dev_net->passives = NULL;
	dev_net->vdcList = NULL;
	dev_net->modelList = NULL;

	
	// Alloc. node and paramerter lists in element first,
	// construct element
	// copy to gpu array
	// set dev_net pointers

	dev_net->n_active = netlist->active_elem.size();
	dev_net->n_nodes = netlist->netNames.size() - 1;
	dev_net->n_passive = netlist->elements.size();
	dev_net->n_vdc = netlist->vdcList.size();

	CUDA_Elem e;
	int e_nodes;
	int e_params;
	int* dev_nodes = NULL;
	float* dev_params = NULL;

	e.model = 0;

	CUDA_Elem* dev_passives = NULL;

	// Passive element list allocation
	
	hipMalloc((void**)&dev_passives, dev_net->n_passive * sizeof(CUDA_Elem));
	dev_net->passives = dev_passives;

	for (int i = 0; i < dev_net->n_passive; i++) {
		// Get array sizes
		e_nodes = netlist->elements[i].nodes.size();
		e_params = netlist->elements[i].params.size();

		// Allocate device element's node and parameter lists
		hipMalloc((void**)&dev_nodes, e_nodes * sizeof(int));
		hipMalloc((void**)&dev_params, e_params * sizeof(float));

		// Copy arrays to GPU
		hipMemcpy(dev_nodes, netlist->elements[i].nodes.data(), e_nodes * sizeof(int), hipMemcpyHostToDevice);
		checkCUDAError("Device element nodelist copy fail!\n");
		hipMemcpy(dev_params, netlist->elements[i].params.data(), e_params * sizeof(float), hipMemcpyHostToDevice);
		checkCUDAError("Device element parameter list copy fail!\n");

		// Set eleement values
		e.nodes = dev_nodes;
		e.params = dev_params;
		e.type = netlist->elements[i].type;

		// Copy element struct to device
		hipMemcpy(dev_passives + i, &e, sizeof(CUDA_Elem), hipMemcpyHostToDevice);
	}
	
}

// Probably should figure out how to ensure all arrays in
// the netlist and elements actually free,
// somehow access or track all these pointers?
void freeGpuNetlist(CUDA_Net* dev_net) {
	//hipFree(dev_net->modelList);
	
	//int n = max(max(netlist->active_elem.size(), netlist->elements.size()), netlist->vdcList.size());
	// Copy Pointers to Structs
	//int numBlocks = ceil(float(n) / 64.0f);

	//dim3 numBlocks3D = dim3(numBlocks, numBlocks, 1);
	//dim3 blockSize = dim3(64.0f, 1, 1);

	//kernFreeArrays << < numBlocks3D, blockSize >> > (n, dev_net);

	// Free Netlist
	if (dev_net->modelList != NULL) hipFree(dev_net->modelList);
	checkCUDAError("Netlist Free Failure!\n");

	if (dev_net->actives != NULL && dev_net->n_active > 0) hipFree(dev_net->actives);
	checkCUDAError("Netlist Free Failure!\n");

	if (dev_net->passives != NULL && dev_net->n_passive > 0) hipFree(dev_net->passives);
	checkCUDAError("Netlist Free Failure!\n");

	if (dev_net->vdcList != NULL && dev_net->n_vdc > 0) hipFree(dev_net->vdcList);
	checkCUDAError("Netlist Free Failure!\n");

	free(dev_net);

}